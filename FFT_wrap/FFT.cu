#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hipfft/hipfft.h>

#include "batchfft.hcu"
#include "uint_util.hcu"

/**
Kernel for performing the FFT wrap

*/
template<class T> __global__ void fft_shift_kernel( hipFloatComplex* data_in, hipFloatComplex* data_out, T dim, unsigned int num_images)
{

	// This is the current pixel number
	unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

	// Number of elements
	unsigned int num_elements = prod(dim);

	// Do nothing if index is out of range
	if( idx < num_elements ){

		//Lets get the coordinates for this pixel
		T src_co = idx_to_co(idx, dim);

		//Where should this data go?
		T dst_co = (src_co+(dim>>1))%dim;

		for( unsigned int i=0; i<num_images; i++ ){
			unsigned int offset = i*num_elements;
			data_out[co_to_idx(dst_co,dim)+offset] = data_in[idx+offset];
		}  
	}
}

/**
Kernel for performing the FFT wrap

This one also permutes the dim_to_traf dimension to the first dimensions for a 1D FFT

*/
template<class T> __global__ void fft_shift_permute_kernel( unsigned int num_elements, hipFloatComplex* data_in, hipFloatComplex* data_out, T dim, T offset, unsigned int dim_to_traf, unsigned int d )
{	
	//This is the current pixel number
	unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

	// Do nothing if index is out of range
	if( idx < num_elements ){

		//Lets get the coordinates for this pixel
		T src_co = idx_to_co(idx, dim)+offset;

		//Where should this data go?
		T dst_co = (src_co+(dim>>1))%dim;
		T new_dim;

		// 'd' is the "direction": 0 or 1, i.e. shift before or after the FFT
		if (d == 1)
		{
			dst_co = shift_down(dst_co,dim_to_traf);
			new_dim = shift_down(dim,dim_to_traf);
		} else {
			dst_co = shift_up(dst_co,dim_to_traf);
			new_dim = shift_up(dim,dim_to_traf);
		}

		data_out[co_to_idx(dst_co,new_dim)] = data_in[idx];
	}
}

template<class T> void 
fft_shift(hipFloatComplex* data_in, hipFloatComplex* data_out, T dim, unsigned int num_images )
{  
	if( data_in == data_out ){
		printf("\nError in fft shifter. Input and output pointer cannot overlap.\n");
		return;
	}

	dim3 blockDim(512,1,1);
	dim3 gridDim((int)ceil((double)prod(dim)/(double)blockDim.x),1,1);

	fft_shift_kernel<T><<< gridDim, blockDim >>>(data_in, data_out, dim, num_images);

	hipError_t err = hipGetLastError();
	if( err != hipSuccess ){
		printf("\nCuda error detected in 'fft_shift': %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
		exit(1);
	}
}


/**
This function will perform the FFT along the dimension specified by dim_to_trafo
in the direction (HIPFFT_FORWARD or HIPFFT_BACKWARD) specified by direction

*/
template<class T> __host__ bool ft_1d(hipFloatComplex* data, T dim, T offset, unsigned int dim_to_traf, int direction, bool do_scale, bool do_shift )
{
	if( dim_to_traf != 1 && !do_shift )
		printf("\nWARNING: 1D FFT of multidimensional dataset WITH NO SHIFT is requested. This is a problem if data is not already permuted!\n");

	unsigned int Nx = ((unsigned int*)&dim)[dim_to_traf];

	hipFloatComplex* temp;  
	dim3 blockDim(512,1, 1);
	dim3 gridDim((int)ceil((double)prod(dim)/(double)blockDim.x),1,1);

	if( do_shift ){
		hipMalloc( (void **) &temp,sizeof(hipFloatComplex)*prod(dim));
		fft_shift_permute_kernel<T><<< gridDim, blockDim >>>(prod(dim), data, (hipFloatComplex*)temp, dim, offset, dim_to_traf, 1 );

		hipError_t err = hipGetLastError();
		if( err != hipSuccess ){
			printf("\nCuda error detected in 'fft_shift_permute_kernel': %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
			exit(1);
		}
	}  
	else
		temp = (hipFloatComplex*) data;

	hipfftHandle plan;
	hipfftResult res;
	
	res = hipfftPlan1d(&plan, Nx, HIPFFT_C2C, prod(dim)/Nx); // Batched 1D FFTs

	if( res != HIPFFT_SUCCESS ){
		printf("\nFATAL ERROR in 'hipfftPlan2d': %s. Quitting.\n", hipGetErrorString(hipGetLastError()));
		exit(1);
	}

	res = hipfftExecC2C(plan, temp, temp, direction);

	if( res != HIPFFT_SUCCESS ){
		printf("\nFATAL ERROR in 'hipfftExecC2C': %s. Quitting.\n", hipGetErrorString(hipGetLastError()));
		exit(1);
	}

	if (direction == HIPFFT_BACKWARD && do_scale)
	{
		hipblasCscal (prod(dim), make_hipFloatComplex(1.0f/Nx,0.0f), (hipFloatComplex*)temp, 1);
	}

	if( do_shift ){
		fft_shift_permute_kernel<T><<< gridDim, blockDim >>>(prod(dim), (hipFloatComplex*)temp, data, shift_down(dim,dim_to_traf), offset, dim_to_traf, 0 );
	
		hipError_t err = hipGetLastError();
		if( err != hipSuccess ){
			printf("\nCuda error detected in 'fft_shift_permute_kernel': %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
			exit(1);
		}

		hipFree(temp);
	}

	res = hipfftDestroy(plan);

	if( res != HIPFFT_SUCCESS ){
		printf("\nFATAL ERROR in 'hipfftDestroy': %s. Quitting.\n", hipGetErrorString(hipGetLastError()));
		exit(1);
	}

	return true;
}

/**
This function determines the correct conversion of the dimension parameters based on the dimensionaly of the problem,
it then calls the ft_1d_wrapper.
*/
bool ft_1d_wrapper(hipFloatComplex* data, uint4 dim, unsigned int dim_to_trans, int direction, bool do_scale, bool do_shift )
{
	uint4 offset = make_uint4(0,0,0,0);

	if (dim.z == 1 && dim.w == 1)
	{
		//One or Two dimensionsal dataset
		ft_1d(data, uint4_to_uint2(dim), uint4_to_uint2(offset), dim_to_trans, direction, do_scale, do_shift );
	}
	else if (dim.w == 1)
	{
		//Three dimensional dataset
		ft_1d(data, uint4_to_uint3(dim),uint4_to_uint3(offset), dim_to_trans, direction, do_scale, do_shift );
	}
	else
	{
		//Four dimensional dataset
		ft_1d(data, dim, offset, dim_to_trans, direction, do_scale, do_shift );
	}

	return true;
}

/**
This wrapper is used for a 2D FFT where both dimensions are transformed.

*/
bool ft_2d_wrapper(hipFloatComplex* data, uint2 dim, int direction, unsigned int num_images, bool do_scale, bool shift)
{
	hipFloatComplex* temp;

	if( shift ){
		hipMalloc( (void **) &temp, sizeof(hipFloatComplex)*prod(dim)*num_images );
		fft_shift<uint2>( data, temp, dim, num_images );
	}
	else
		temp = (hipFloatComplex*)data;

	if( num_images == 1 ){

		// Single FFT

		hipfftHandle plan;
		hipfftResult res;
		
		res = hipfftPlan2d(&plan, dim.y, dim.x, HIPFFT_C2C);

		if( res != HIPFFT_SUCCESS ){
			printf("\nFATAL ERROR in 'hipfftPlan2d': %s. Quitting.\n", hipGetErrorString(hipGetLastError()));
			exit(1);
		}

		res = hipfftExecC2C(plan, temp, temp, direction);

		if( res != HIPFFT_SUCCESS ){
			printf("\nFATAL ERROR in 'hipfftExecC2C': %s. Quitting.\n", hipGetErrorString(hipGetLastError()));
			exit(1);
		}

		res = hipfftDestroy(plan);

		if( res != HIPFFT_SUCCESS ){
			printf("\nFATAL ERROR in 'hipfftDestroy': %s. Quitting.\n", hipGetErrorString(hipGetLastError()));
			exit(1);
		}
	}
	else{

		// Batched 2D FFTs

		batchfftHandle batchplan;
		hipfftResult res;
		
		res = batchfftPlan2d(&batchplan, dim.y, dim.x, HIPFFT_C2C, num_images);

		if( res != HIPFFT_SUCCESS ){
			printf("\nFATAL ERROR in 'batchfftPlan2d': %s. Quitting.\n", hipGetErrorString(hipGetLastError()));
			exit(1);
		}

		res = batchfftExecute(batchplan, temp, temp, direction);

		if( res != HIPFFT_SUCCESS ){
			printf("\nFATAL ERROR in 'batchfftExecute': %s. Quitting.\n", hipGetErrorString(hipGetLastError()));
			exit(1);
		}

		res = batchfftDestroy(&batchplan);

		if( res != HIPFFT_SUCCESS ){
			printf("\nFATAL ERROR in 'batchfftDestroy': %s. Quitting.\n", hipGetErrorString(hipGetLastError()));
			exit(1);
		}
	}

	if (direction == HIPFFT_BACKWARD && do_scale)
	{
		hipblasCscal( prod(dim)*num_images, make_hipFloatComplex(1.0f/prod(dim),0.0f), (hipFloatComplex*)temp, 1 );
	}

	if( shift ){
		fft_shift<uint2>( (hipFloatComplex*)temp, data, dim, num_images );
		hipFree( temp );
	}

	return true;
}

/** 
This function is used for a 3D FFT where all dimensions are transformed

*/
bool ft_3d_wrapper(hipFloatComplex* data, uint3 dim, int direction, unsigned int num_images, bool do_scale, bool shift)
{
	hipFloatComplex* temp;

	if(shift){
		hipMalloc( (void **) &temp, sizeof(hipFloatComplex)*prod(dim)*num_images );
		fft_shift<uint3>(data, (hipFloatComplex*)temp, dim, num_images );
	}
	else
		temp = (hipFloatComplex*)data;

	if( num_images == 1 ){

		// Single FFT

		hipfftHandle plan;
		hipfftResult res;
		
		res = hipfftPlan3d(&plan, dim.z, dim.y, dim.x, HIPFFT_C2C);

		if( res != HIPFFT_SUCCESS ){
			printf("\nFATAL ERROR in 'hipfftPlan3d': %s. Quitting.\n", hipGetErrorString(hipGetLastError()));
			exit(1);
		}

		res = hipfftExecC2C(plan, temp, temp, direction);

		if( res != HIPFFT_SUCCESS ){
			printf("\nFATAL ERROR in 'hipfftExecC2C': %s. Quitting.\n", hipGetErrorString(hipGetLastError()));
			exit(1);
		}

		res = hipfftDestroy(plan);

		if( res != HIPFFT_SUCCESS ){
			printf("\nFATAL ERROR in 'hipfftDestroy': %s. Quitting.\n", hipGetErrorString(hipGetLastError()));
			exit(1);
		}
	}
	else{

		// Batched 3D FFTs
		batchfftHandle batchplan;
		hipfftResult res;
		
		res = batchfftPlan3d(&batchplan, dim.z, dim.y, dim.x, HIPFFT_C2C, num_images);

		if( res != HIPFFT_SUCCESS ){
			printf("\nFATAL ERROR in 'batchfftPlan3d': %s. Quitting.\n", hipGetErrorString(hipGetLastError()));
			exit(1);
		}

		res = batchfftExecute(batchplan, temp, temp, direction);

		if( res != HIPFFT_SUCCESS ){
			printf("\nFATAL ERROR in 'batchfftExecute': %s. Quitting.\n", hipGetErrorString(hipGetLastError()));
			exit(1);
		}

		res = batchfftDestroy(&batchplan);

		if( res != HIPFFT_SUCCESS ){
			printf("\nFATAL ERROR in 'batchfftDestroy': %s. Quitting.\n", hipGetErrorString(hipGetLastError()));
			exit(1);
		}
	}

	if (direction == HIPFFT_BACKWARD && do_scale)
	{
		hipblasCscal (prod(dim)*num_images, make_hipFloatComplex(1.0f/prod(dim),0.0f), (hipFloatComplex*)temp, 1);
	}

	if(shift){
		fft_shift<uint3>( (hipFloatComplex*)temp, data, dim, num_images );
		hipFree( temp );
	}


	return true;
}

/**
This is the exported function, which can be called from the main program.
It calls the wrapper.

It performs the FFT from image space to k-space (i.e. forward transform)

*/
__host__ bool I2K(hipFloatComplex* data, uint4 dim, unsigned int dim_to_trans, bool do_scale, bool do_shift )
{
	return ft_1d_wrapper(data,dim,dim_to_trans, HIPFFT_FORWARD, do_scale, do_shift );
}


/**
This is the exported function, which can be called from the main program.
It calls the wrapper.

It performs the INVERSE FFT from k-space to image space (inverse transform)

*/
__host__ bool K2I(hipFloatComplex* data, uint4 dim, unsigned int dim_to_trans, bool do_scale, bool do_shift )
{
	return ft_1d_wrapper(data,dim,dim_to_trans, HIPFFT_BACKWARD, do_scale, do_shift );
}

/**
Transform of all relevant dimensions from image to k space.

Calls the appropriate wrapper.

*/
__host__ bool I2K_ALL(hipFloatComplex* data, uint4 dim, unsigned int num_images, bool do_scale, bool do_shift)
{
	if (dim.z == 1 && dim.w == 1)
	{
		//One or Two dimensionsal dataset
		ft_2d_wrapper(data, uint4_to_uint2(dim), HIPFFT_FORWARD, num_images, do_scale, do_shift);
	}
	else if (dim.w == 1)
	{
		//Three dimensional dataset
		ft_3d_wrapper(data, uint4_to_uint3(dim), HIPFFT_FORWARD, num_images, do_scale, do_shift);
	}
	else
	{
		if( num_images>1 ){
			printf("\nI2K_ALL cannot handles batches at the moment!. Quitting.\n");
			exit(1);
		}

		//Four dimensional dataset
		//The CUFFT library doesn't support 4D transforms, so we'll have to hack it
		for (unsigned int i = 0; i < dim.w; i++)
		{
			ft_3d_wrapper( (data+i*prod(uint4_to_uint3(dim))), uint4_to_uint3(dim), HIPFFT_FORWARD, num_images, do_scale, do_shift );
		}
		ft_1d_wrapper( data,dim,3,HIPFFT_FORWARD, do_scale, do_shift ); //Last dimension manually
	}

	return true;
}

__host__ bool I2K_ALL(hipFloatComplex* data, uint3 dim, unsigned int num_images, bool do_scale, bool do_shift)
{
	if (dim.z == 1 )
	{
		//One or Two dimensionsal dataset
		ft_2d_wrapper(data, uint3_to_uint2(dim),HIPFFT_FORWARD, num_images, do_scale, do_shift);
	}
	else
	{
		//Three dimensional dataset
		ft_3d_wrapper(data, dim, HIPFFT_FORWARD, num_images, do_scale, do_shift);
	}

	return true;
}

__host__ bool I2K_ALL(hipFloatComplex* data, uint2 dim, unsigned int num_images, bool do_scale, bool do_shift)
{
	//One or Two dimensionsal dataset
	ft_2d_wrapper(data, dim, HIPFFT_FORWARD, num_images, do_scale, do_shift);

	return true;
}

/**
Transform of all relevant dimensions from k-space to image space.

Calls the appropriate wrapper.

*/
__host__ bool K2I_ALL(hipFloatComplex* data, uint4 dim, unsigned int num_images, bool do_scale, bool do_shift)
{
	if (dim.z == 1 && dim.w == 1)
	{
		//One or Two dimensionsal dataset
		ft_2d_wrapper(data, uint4_to_uint2(dim),HIPFFT_BACKWARD, num_images, do_scale, do_shift);
	}
	else if (dim.w == 1)
	{
		//Three dimensional dataset
		ft_3d_wrapper(data, uint4_to_uint3(dim), HIPFFT_BACKWARD, num_images, do_scale, do_shift);
	}
	else
	{

		if( num_images>1 ){
			printf("\nK2I_ALL cannot handles batches at the moment!. Quitting.\n");
			exit(1);
		}

		//Four dimensional dataset
		//The CUFFT library doesn't support 4D transforms, so we'll have to hack it
		for (unsigned int i = 0; i < dim.w; i++)
		{
			ft_3d_wrapper((data+i*prod(uint4_to_uint3(dim))), uint4_to_uint3(dim), HIPFFT_BACKWARD, num_images, do_scale, do_shift);
		}
		ft_1d_wrapper( data, dim, 3, HIPFFT_BACKWARD, do_scale, do_shift ); //Last dimension manually
	}

	return true;
}

__host__ bool K2I_ALL(hipFloatComplex* data, uint3 dim, unsigned int num_images, bool do_scale, bool do_shift)
{
	if (dim.z == 1 )
	{
		//One or Two dimensionsal dataset
		ft_2d_wrapper(data, uint3_to_uint2(dim), HIPFFT_BACKWARD, num_images, do_scale, do_shift);
	}
	else
	{
		//Three dimensional dataset
		ft_3d_wrapper(data, dim, HIPFFT_BACKWARD, num_images, do_scale, do_shift);
	}

	return true;
}

__host__ bool K2I_ALL(hipFloatComplex* data, uint2 dim, unsigned int num_images, bool do_scale, bool do_shift)
{
	//One or Two dimensionsal dataset
	ft_2d_wrapper(data, dim, HIPFFT_BACKWARD, num_images, do_scale, do_shift);

	return true;
}

// template instantiation

template void fft_shift(hipFloatComplex*, hipFloatComplex*, uint2, unsigned int );
template void fft_shift(hipFloatComplex*, hipFloatComplex*, uint3, unsigned int );
template void fft_shift(hipFloatComplex*, hipFloatComplex*, uint4, unsigned int );
