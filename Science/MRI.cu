#include "hip/hip_runtime.h"
// hello world

#include "MRI.hcu"
//#include "float_util.hcu"
#include <stdio.h>

#include <Meta/CUDA.h>
__constant__ float3 b;
__constant__ float gx;
__constant__ float gy;
//__constant__ float gy;

//__constant__ float flip;

const float T1 = 1e-5; // spin lattice in seconds.
const float T2 = 1e-6; // spin spin in seconds.
const float GYROMAGNETIC_RATIO = 42.58e6; // hertz pr tesla
// const float BOLTZMANN_CONSTANT = 1.3805e-23; // Joule / Kelvin
// const float PLANCK_CONSTANT = 6.626e-34; // Joule * seconds


struct mat3x3 {
    float3 r1;
    float3 r2;
    float3 r3;
};

// __host__ __device__ float3 getRow(mat3x3 m, unsigned int i) {
//    switch (i) {        
//    case 0: return r1;
//    case 1: return r2;
//    case 2: return r3;
//    }
//    return make_float3(0.0,0.0,0.0);
// }

// ----- reduction ----

template<class T>
struct SharedMemory
{
    __device__ inline operator       T*()
    {
        extern __shared__ int __smem[];
        return (T*)__smem;
    }

    __device__ inline operator const T*() const
    {
        extern __shared__ int __smem[];
        return (T*)__smem;
    }
};


template <class T>
__global__ void
reduce3(T *g_idata, T *g_odata, unsigned int n, T zero)
{
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

    T mySum = (i < n) ? g_idata[i] : zero;
    if (i + blockDim.x < n) 
        mySum += g_idata[i+blockDim.x];  

    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    for(unsigned int s=blockDim.x/2; s>0; s>>=1) 
    {
        if (tid < s) 
        {
            sdata[tid] = mySum = mySum + sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem 
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}
// -------------------------------



__host__ __device__ float3 getCol1(mat3x3 m) {

    return make_float3(m.r1.x, m.r2.x, m.r3.x);
}

__host__ __device__ float3 getCol2(mat3x3 m) {

    return make_float3(m.r1.y, m.r2.y, m.r3.y);
}

__host__ __device__ float3 getCol3(mat3x3 m) {

    return make_float3(m.r1.z, m.r2.z, m.r3.z);
}


__host__ __device__ mat3x3 make_mat3x3(float3 r1, float3 r2, float3 r3) {
    mat3x3 m;
    m.r1 = r1;
    m.r2 = r2;
    m.r3 = r3;
    return m;
}

__host__ __device__ mat3x3 operator*(mat3x3 m1, mat3x3 m2) {
    return make_mat3x3(make_float3(dot(m1.r1, getCol1(m2)), dot(m1.r1, getCol2(m2)), dot(m1.r1, getCol3(m2))),
                       make_float3(dot(m1.r2, getCol1(m2)), dot(m1.r2, getCol2(m2)), dot(m1.r2, getCol3(m2))),
                       make_float3(dot(m1.r3, getCol1(m2)), dot(m1.r3, getCol2(m2)), dot(m1.r3, getCol3(m2)))
                       );
}

__host__ __device__ float3 operator*(mat3x3 m, float3 v) {
    return make_float3(dot(m.r1, v), 
                       dot(m.r2, v), 
                       dot(m.r3, v));
}

__host__ __device__ mat3x3 rotX(float angle) {
    return make_mat3x3(make_float3(1.0,        0.0, 0.0       ),
                       make_float3(0.0, cos(angle), sin(angle)),
                       make_float3(0.0,-sin(angle), cos(angle))
                       );
}

__host__ __device__ mat3x3 rotZ(float angle) {
    return make_mat3x3(make_float3( cos(angle), sin(angle), 0.0),
                       make_float3(-sin(angle), cos(angle), 0.0),
                       make_float3(        0.0,        0.0, 1.0));
}

__host__ __device__ mat3x3 relax(float dt, float t1, float t2) {
    return make_mat3x3(make_float3(1.0/exp(dt/t2),         0.0,            0.0),
                       make_float3(        0.0, 1.0/exp(dt/t2),            0.0),
                       make_float3(        0.0,         0.0, 1.0-(1.0/exp(dt/t1)))
                       );
}

__host__ __device__ mat3x3 rf(float phaseAngle, float flipAngle) {
    return rotZ(phaseAngle) * rotX(flipAngle) * rotZ(-phaseAngle);
}

void MRI_test(hipFloatComplex* input) {
        
}

__global__ void MRI_step_kernel(float dt, float3* lab_spins, float3* ref_spins, SpinProperty* props, unsigned int size, float thetime) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= size)
        return;
    // lab_spins[idx] = make_float3(gx,gx,gx);
    // return;

    float omega = GYROMAGNETIC_RATIO * b.z;
    float3 m = ref_spins[idx];
    float dtt1 = dt/props[idx].t1;
    float dtt2 = dt/props[idx].t2;

    float posX = float(idx % 600);
    float posY = float(idx / 600);
    

    // fid rotation
    m = rotX(b.x)*m;
    // relaxation
    m += make_float3(-m.x*dtt2, -m.y*dtt2, (props[idx].eq-m.z)*dtt1);
    ref_spins[idx] = m;
    // gradient rotation
    m = rotZ(GYROMAGNETIC_RATIO*gx*posX*dt)*m;
    m = rotZ(GYROMAGNETIC_RATIO*gy*posY*dt)*m;
    // m = rotZ(gx)*m;

    // reference to laboratory
    lab_spins[idx] = make_float3(m.x * cos(omega * thetime) - m.y * sin(omega*thetime), m.x * sin(omega * thetime) + m.y * cos(omega*thetime),  m.z);



    // lab_spins[idx] += dt * (cross(GYROMAGNETIC_RATIO * m, b) - make_float3(m.x / T2, m.y / T2, 0.0) - make_float3(0.0, 0.0, (eq[idx] - m.z) / T1));
    
    //lab_spins[idx] = (rotZ(GYROMAGNETIC_RATIO * dt) * relax(dt, T1, T2)) * m;
    // m = relax(dt,T1,T2) * m;
    //lab_spins[idx] = relax(dt,T1,T2) * m;
    //lab_spins[idx] = /*(rotZ(GYROMAGNETIC_RATIO * 0.0 * dt) * relax(dt, T1, T2) * rf(0.0, b.x))*/  rotX(b.x) * m;
    //lab_spins[idx] = /*rotZ(GYROMAGNETIC_RATIO * b.z * dt) **/ (relax(dt, T1, T2) * (rotX(b.x) * m));
}

__global__ void MRI_step_kernel_anal(float t, float3* spin_packs, float* eq, unsigned int size) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= size)
        return;
        
    float3 m = spin_packs[idx];

    float larmorFrequency = GYROMAGNETIC_RATIO;

    float2 e1;
    e1.x = cos(larmorFrequency * t);
    e1.y = sin(larmorFrequency * t);
    float2 e2;
    e2.x = -sin(larmorFrequency * t);
    e2.y = cos(larmorFrequency * t);


    float T1exp = exp(-t/T1);
    float T2exp = exp(-t/T2);

    float3 m0 = make_float3(0,  eq[idx], 0);


    float3 localNetMagnetization;

    float meq = m0.y; // HAck, Meq is the size of the default field.

    localNetMagnetization.x = T2exp * m0.x;
    localNetMagnetization.y = T2exp * m0.y;
    localNetMagnetization.z = m0.z * T1exp + meq * 1 * (1-T1exp);

    m.x = localNetMagnetization.x * e1.x + localNetMagnetization.y * e2.x;
    m.y = localNetMagnetization.x * e1.y + localNetMagnetization.y * e2.y;
    m.z = localNetMagnetization.z;


    //m = localNetMagnetization;

    spin_packs[idx] = m;
}



__host__ void printVec3(float3 f) {
    printf("[%f %f %f]\n",f.x, f.y, f.z);
}

__host__ void printMat(mat3x3 m) {
    printVec3(m.r1);
    printVec3(m.r2);
    printVec3(m.r3);
}

__host__ void printFloat(float f) {
    printf("%f\n", f);
}

float thetime = 0.0;

__host__ float3 MRI_step(float dt, float3* lab_spins, float3* ref_spins,
                       SpinProperty* props, unsigned int w, unsigned int h, float3 _b, float _gx, float _gy) {
    hipMemcpyToSymbol(HIP_SYMBOL(b), &_b, sizeof(float3));
    hipMemcpyToSymbol(HIP_SYMBOL(gx), &_gx, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(gy), &_gy, sizeof(float));

	dim3 blockDim(256,1,1);
	dim3 gridDim(int(((double)(w*h))/(double)blockDim.x),1,1);

    /* printf("time = %e\n",dt); */

    //MRI_step_kernel_anal<<< gridDim, blockDim >>>(dt, (float3*)spin_packs, eq, w*h);
    //    MRI_step_kernel<<< gridDim, blockDim >>>(dt, (float3*)spin_packs, eq, w*h);
    
    float3* odata;

    int reduceBlocks = 256;

    hipMalloc((void**)&odata, reduceBlocks * sizeof(float3));

    thetime += dt;
    // MRI_step_kernel_anal<<< gridDim, blockDim >>>(thetime, (float3*)spin_packs, eq, w*h);
    MRI_step_kernel<<< gridDim, blockDim >>>(dt, lab_spins, ref_spins, props, w*h, thetime);

    CHECK_FOR_CUDA_ERROR();
    hipDeviceSynchronize();

    // printf("gx = ");
    // printFloat(_gx);

    hipMemcpy(odata, lab_spins, reduceBlocks*sizeof(float3),hipMemcpyDeviceToDevice);
    reduce3<float3><<< gridDim, blockDim >>>(lab_spins, odata, w*h, make_float3(0,0,0));

    hipDeviceSynchronize();

    float3* c_odata = (float3*)malloc(reduceBlocks*sizeof(float3));


    hipMemcpy( c_odata, odata, reduceBlocks * sizeof(float3), hipMemcpyDeviceToHost);
    float3 gpu_result = make_float3(0,0,0);
    for(int i=0; i < reduceBlocks; i++) 
        {
            gpu_result += c_odata[i];
        }

    hipFree(odata);

    //gpu_result /= w*h;

    /* printf("reduced = "); */
    /* printVec3(gpu_result); */
    float3 v = make_float3(0.8, 0.1, 0.1);
    /* printf("v = "); */
    /* printVec3(v); */

    /* printf("relax = "); */

    /* printMat(relax(dt, T1, T2)); */

    float3 v2 = relax(dt, T1, T2) * v;

    /* printf(" v * relax = "); */

    /* printVec3(v2); */

    return gpu_result;
}
